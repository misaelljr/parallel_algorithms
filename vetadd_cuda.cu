#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

__global__
void vectAdd(int *A, int *B, int *C)
{
	int i = blockDim.x * threadIdx.x + blockIdx.x;

	if(i < N)
		C[i] = A[i] + B[i];
}

int main(int argc, char const *argv[])
{
	int A[N], B[N], C[N];

	int size = N * sizeof(int);

	for (int i = 0; i < N; i++)
	{
		A[i] = i; 
	}

	for (int i = 0; i < N; i++)
	{
		B[i] = i; 
	}

	int *d_A, *d_B, *d_C;

	hipMalloc((void **)&d_A, N*sizeof(int));
	hipMemcpy(d_A, A, N*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, N*sizeof(int));
	hipMemcpy(d_B, B, N*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void **)&d_C, size);

	vectAdd<<<N, 1>>>(d_A, d_B, d_C);

	hipMemcpy(C, d_C, N*sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i<N; i++) {
        printf("%d\n", C[i]);
    }

    hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
